#include "hip/hip_runtime.h"
/*
  Copyright (C) 2014,2015,2016 The ESPResSo project
  
  This file is part of ESPResSo.
  
  ESPResSo is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

#include "EspressoSystemInterface.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#ifdef BARNES_HUT
#include "actor/DipolarBarnesHut_cuda.cuh"
#endif

// These functions will split the paritlce data structure into individual arrays for each property

// Position and charge
#ifndef BARNES_HUT
__global__ void split_kernel_rq(CUDA_particle_data *particles, float *r, float *q, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  r[3*idx + 0] = p.p[0];
  r[3*idx + 1] = p.p[1];
  r[3*idx + 2] = p.p[2];
  #ifdef ELECTROSTATICS
  q[idx] = p.q;
  #endif
}
#else
__global__ void split_kernel_rq(CUDA_particle_data *particles, float *rx, float *ry, float *rz, float *q, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  rx[idx] = p.p[0];
  ry[idx] = p.p[1];
  rz[idx] = p.p[2];
  #ifdef ELECTROSTATICS
  q[idx] = p.q;
  #endif
}
#endif // BARNES_HUT

// Charge only
__global__ void split_kernel_q(CUDA_particle_data *particles,float *q, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

#ifdef ELECTROSTRATICS
  CUDA_particle_data p = particles[idx];

  q[idx] = p.q;
#endif
}

// Position only
#ifndef BARNES_HUT
__global__ void split_kernel_r(CUDA_particle_data *particles, float *r, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  r[idx + 0] = p.p[0];
  r[idx + 1] = p.p[1];
  r[idx + 2] = p.p[2];
}
#else
__global__ void split_kernel_r(CUDA_particle_data *particles, float *rx, float *ry, float *rz, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  rx[idx] = p.p[0];
  ry[idx] = p.p[1];
  rz[idx] = p.p[2];
}
#endif // BARNES_HUT

// Velocity
__global__ void split_kernel_v(CUDA_particle_data *particles, float *v, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  v[idx + 0] = p.v[0];
  v[idx + 1] = p.v[1];
  v[idx + 2] = p.v[2];
}


#ifdef DIPOLES
// Dipole moment
#ifndef BARNES_HUT
__global__ void split_kernel_dip(CUDA_particle_data *particles, float *dip, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  dip[idx + 0] = p.dip[0];
  dip[idx + 1] = p.dip[1];
  dip[idx + 2] = p.dip[2];
}
#else
__global__ void split_kernel_dip(CUDA_particle_data *particles, float *dipx, float *dipy, float *dipz, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  dipx[idx] = p.dip[0];
  dipy[idx] = p.dip[1];
  dipz[idx] = p.dip[2];
}
#endif // BARNES_HUT
#endif // DIPOLES

__global__ void split_kernel_quatu(CUDA_particle_data *particles, float *quatu, int n) {
#ifdef ROTATION
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  quatu[idx + 0] = p.quatu[0];
  quatu[idx + 1] = p.quatu[1];
  quatu[idx + 2] = p.quatu[2];
#endif
}

void EspressoSystemInterface::reallocDeviceMemory(int n) {

#ifdef BARNES_HUT

  if ((n != m_gpu_npart) || (m_blocks == 0) || (m_bhnnodes == 0))
  {
	  hipDeviceProp_t deviceProp;
	  hipGetDeviceProperties(&deviceProp, 0); // TODO: local MPI node "dev" value should be here

	  m_blocks = deviceProp.multiProcessorCount;
	  m_bhnnodes = n * 8;
	  if (m_bhnnodes < 1024 * m_blocks) m_bhnnodes = 1024 * m_blocks;
	  while ((m_bhnnodes & (WARPSIZE - 1)) != 0) m_bhnnodes++;
	  m_bhnnodes--;
	  //srand(time(NULL));
  }

  if (m_arrl.err == 0) cuda_safe_mem(hipMalloc((void **)&m_arrl.err, sizeof(int)));

  if ((m_arrl.child == 0) || (n != m_gpu_npart))
  {
	  if (m_arrl.child != 0) cuda_safe_mem(hipFree(m_arrl.child));
	  cuda_safe_mem(hipMalloc((void **)&m_arrl.child, sizeof(int) * (m_bhnnodes + 1) * 8));
  }

  if ((m_arrl.count == 0) || (n != m_gpu_npart))
  {
   if (m_arrl.count != 0) cuda_safe_mem(hipFree(m_arrl.count));
   cuda_safe_mem(hipMalloc((void **)&m_arrl.count, sizeof(int) * (m_bhnnodes + 1)));
  }

  if ((m_arrl.start == 0) || (n != m_gpu_npart))
  {
   if (m_arrl.start != 0) cuda_safe_mem(hipFree(m_arrl.start));
   cuda_safe_mem(hipMalloc((void **)&m_arrl.start, sizeof(int) * (m_bhnnodes + 1)));
  }

  if ((m_arrl.sort == 0) || (n != m_gpu_npart))
  {
   if (m_arrl.sort != 0) cuda_safe_mem(hipFree(m_arrl.sort));
   cuda_safe_mem(hipMalloc((void **)&m_arrl.sort, sizeof(int) * (m_bhnnodes + 1)));
  }

  if ((m_mass == 0) || (n != m_gpu_npart))
  {
   if (m_mass != 0) cuda_safe_mem(hipFree(m_mass));
   cuda_safe_mem(hipMalloc((void **)&m_mass, sizeof(float) * (m_bhnnodes + 1)));

   float *mass = new float [n];
   for(int i = 0; i < n; i++) {
   		mass[i] = 1.0f;
   }
   cuda_safe_mem(hipMemcpy(m_mass, mass, sizeof(float) * n, hipMemcpyHostToDevice));
   delete[] mass;
  }

  if ((m_boxl.maxx == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.maxx != 0) cuda_safe_mem(hipFree(m_boxl.maxx));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.maxx, sizeof(float) * m_blocks * 3));
  }

  if ((m_boxl.maxy == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.maxy != 0) cuda_safe_mem(hipFree(m_boxl.maxy));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.maxy, sizeof(float) * m_blocks * 3));
  }

  if ((m_boxl.maxz == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.maxz != 0) cuda_safe_mem(hipFree(m_boxl.maxz));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.maxz, sizeof(float) * m_blocks * 3));
  }

  if ((m_boxl.minx == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.minx != 0) cuda_safe_mem(hipFree(m_boxl.minx));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.minx, sizeof(float) * m_blocks * 3));
  }

  if ((m_boxl.miny == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.miny != 0) cuda_safe_mem(hipFree(m_boxl.miny));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.miny, sizeof(float) * m_blocks * 3));
  }

  if ((m_boxl.minz == 0) || (n != m_gpu_npart))
  {
   if (m_boxl.minz != 0) cuda_safe_mem(hipFree(m_boxl.minz));
   cuda_safe_mem(hipMalloc((void **)&m_boxl.minz, sizeof(float) * m_blocks * 3));
  }
#endif // BARNES_HUT

#ifndef BARNES_HUT
  if(m_needsRGpu && ((n != m_gpu_npart) || (m_r_gpu_begin == 0))) {
    if(m_r_gpu_begin != 0)
      cuda_safe_mem(hipFree(m_r_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_r_gpu_begin, 3*n*sizeof(float)));
    m_r_gpu_end = m_r_gpu_begin + 3*n;
#else
  if(m_needsRGpu && ( (n != m_gpu_npart) || (m_rx_gpu_begin == 0) || (m_ry_gpu_begin == 0) || (m_rz_gpu_begin == 0) )) {
	if(m_rx_gpu_begin != 0) cuda_safe_mem(hipFree(m_rx_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_rx_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_rx_gpu_end = m_rx_gpu_begin + m_bhnnodes + 1;

    if(m_ry_gpu_begin != 0) cuda_safe_mem(hipFree(m_ry_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_ry_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_ry_gpu_end = m_ry_gpu_begin + m_bhnnodes + 1;

    if(m_rz_gpu_begin != 0) cuda_safe_mem(hipFree(m_rz_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_rz_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_rz_gpu_end = m_rz_gpu_begin + m_bhnnodes + 1;
#endif
  }
#ifdef DIPOLES
#ifndef BARNES_HUT
  if(m_needsDipGpu && ((n != m_gpu_npart) || (m_dip_gpu_begin == 0))) {
    if(m_dip_gpu_begin != 0)
      cuda_safe_mem(hipFree(m_dip_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_dip_gpu_begin, 3*n*sizeof(float)));
    m_dip_gpu_end = m_dip_gpu_begin + 3*n;
#else
  if(m_needsDipGpu && ((n != m_gpu_npart) || (m_dipx_gpu_begin == 0) || (m_dipy_gpu_begin == 0) || (m_dipz_gpu_begin == 0))) {
    if(m_dipx_gpu_begin != 0) cuda_safe_mem(hipFree(m_dipx_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_dipx_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_ux_gpu_end = m_ux_gpu_begin + m_bhnnodes + 1;

    if(m_dipy_gpu_begin != 0) cuda_safe_mem(hipFree(m_dipy_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_dipy_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_uy_gpu_end = m_uy_gpu_begin + m_bhnnodes + 1;

    if(m_dipz_gpu_begin != 0) cuda_safe_mem(hipFree(m_dipz_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_dipz_gpu_begin, (m_bhnnodes + 1) * sizeof(float)));
    //m_uz_gpu_end = m_uz_gpu_begin + m_bhnnodes + 1;
#endif // BARNES_HUT
  }
#endif // DIPOLES
  if(m_needsVGpu && ((n != m_gpu_npart) || (m_v_gpu_begin == 0))) {
    if(m_v_gpu_begin != 0)
      cuda_safe_mem(hipFree(m_v_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_v_gpu_begin, 3*n*sizeof(float)));
    m_v_gpu_end = m_v_gpu_begin + 3*n;
  }

  if(m_needsQGpu && ((n != m_gpu_npart) || (m_q_gpu_begin == 0))) {
    if(m_q_gpu_begin != 0)
      cuda_safe_mem(hipFree(m_q_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_q_gpu_begin, 3*n*sizeof(float)));
    m_q_gpu_end = m_q_gpu_begin + 3*n;
  }

  if(m_needsQuatuGpu && ((n != m_gpu_npart) || (m_quatu_gpu_begin == 0))) {
    if(m_quatu_gpu_begin != 0)
      cuda_safe_mem(hipFree(m_quatu_gpu_begin));
    cuda_safe_mem(hipMalloc(&m_quatu_gpu_begin, 3*n*sizeof(float)));
    m_quatu_gpu_end = m_quatu_gpu_begin + 3*n;
  }
  m_gpu_npart = n;

#ifdef BARNES_HUT
  fillConstantPointers(this->rxGpuBegin(), this->ryGpuBegin(), this->rzGpuBegin(),
  		this->dipxGpuBegin(), this->dipyGpuBegin(), this->dipzGpuBegin(),
  		this->npart_gpu(), this->bhnnodes(), this->arrl(), this->boxl(), this->massGpuBegin());
  initBH(this->blocksGpu());
#endif
}

void EspressoSystemInterface::split_particle_struct() {
  int n = gpu_get_global_particle_vars_pointer_host()->number_of_particles;
  if(n == 0) 
    return;

  ESIF_TRACE(printf("n = %d, m_gpu_npart = %d\n", n, m_gpu_npart));
    
  dim3 grid(n/512+1,1,1);
  dim3 block(512,1,1);

  if(m_needsQGpu && !m_needsRGpu)
      split_kernel_q<<<grid,block>>>(gpu_get_particle_pointer(), m_q_gpu_begin,n);

#ifndef BARNES_HUT
  if(m_needsQGpu && m_needsRGpu)
    split_kernel_rq<<<grid,block>>>(gpu_get_particle_pointer(), m_r_gpu_begin,m_q_gpu_begin,n);
  if(!m_needsQGpu && m_needsRGpu)
    split_kernel_r<<<grid,block>>>(gpu_get_particle_pointer(), m_r_gpu_begin,n);
#ifdef DIPOLES
  if(m_needsDipGpu)
    split_kernel_dip<<<grid,block>>>(gpu_get_particle_pointer(), m_dip_gpu_begin,n);
#endif // DIPOLES
#else
  if(m_needsQGpu && m_needsRGpu)
    split_kernel_rq<<<grid,block>>>(gpu_get_particle_pointer(), m_rx_gpu_begin, m_ry_gpu_begin, m_rz_gpu_begin, m_q_gpu_begin, n);
  if(!m_needsQGpu && m_needsRGpu)
    split_kernel_r<<<grid,block>>>(gpu_get_particle_pointer(), m_rx_gpu_begin, m_ry_gpu_begin, m_rz_gpu_begin, n);
#ifdef DIPOLES
  if(m_needsDipGpu)
    split_kernel_dip<<<grid,block>>>(gpu_get_particle_pointer(), m_dipx_gpu_begin, m_dipy_gpu_begin, m_dipz_gpu_begin, n);
#endif // DIPOLES

#endif // BARNES_HUT

  if(m_needsVGpu)
        split_kernel_v<<<grid,block>>>(gpu_get_particle_pointer(), m_v_gpu_begin,n);

  if(m_needsQuatuGpu)
    split_kernel_quatu<<<grid,block>>>(gpu_get_particle_pointer(), m_quatu_gpu_begin,n);
}
